#include <iostream>
#include <fstream>
#include <vector>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 256

__global__ void movingAverageKernel(float* input, float* output, int N, int windowSize) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        float sum = 0;
        int count = 0;
        for (int i = idx - windowSize / 2; i <= idx + windowSize / 2; ++i) {
            if (i >= 0 && i < N) {
                sum += input[i];
                count++;
            }
        }
        output[idx] = sum / count;
    }
}

std::vector<float> readCSV(const std::string& filename) {
    std::ifstream file(filename);
    std::vector<float> data;
    std::string line;
    while (std::getline(file, line)) {
        data.push_back(std::stof(line));
    }
    return data;
}

void writeCSV(const std::string& filename, const std::vector<float>& data) {
    std::ofstream file(filename);
    for (const auto& val : data) {
        file << val << "\n";
    }
}

int main(int argc, char* argv[]) {
    if (argc != 4) {
        std::cerr << "Usage: ./main <input.csv> <output.csv> <window_size>\n";
        return 1;
    }

    std::string inputFile = argv[1];
    std::string outputFile = argv[2];
    int windowSize = std::stoi(argv[3]);

    std::vector<float> input = readCSV(inputFile);
    int N = input.size();

    float *d_input, *d_output;
    hipMalloc(&d_input, N * sizeof(float));
    hipMalloc(&d_output, N * sizeof(float));

    hipMemcpy(d_input, input.data(), N * sizeof(float), hipMemcpyHostToDevice);

    int numBlocks = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
    movingAverageKernel<<<numBlocks, BLOCK_SIZE>>>(d_input, d_output, N, windowSize);
    hipDeviceSynchronize();

    std::vector<float> output(N);
    hipMemcpy(output.data(), d_output, N * sizeof(float), hipMemcpyDeviceToHost);

    writeCSV(outputFile, output);

    hipFree(d_input);
    hipFree(d_output);

    std::cout << "Processed " << N << " signal points with window size " << windowSize << ".\n";
    return 0;
}
